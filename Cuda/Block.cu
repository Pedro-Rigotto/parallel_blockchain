#include "hip/hip_runtime.h"
#define NUM_THREADS 384
#define MAX_NONCE 100000000

#include "Block.cuh"
#include "sha256.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <sstream>

Block::Block(uint32_t nIndexIn, const std::string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _nNonce = 0;
    _tTime = time(nullptr);

    sHash = _CalculateHash();
}

__device__ unsigned int validHashFound = 0;

__device__ void uint32ToString(uint32_t value, char* buffer) {
    int i = 0;
    do {
        buffer[i++] = (value % 10) + '0';
        value /= 10;
    } while (value != 0);
    buffer[i] = '\0';
    
    // Reverter a string
    for (int j = 0; j < i / 2; ++j) {
        char temp = buffer[j];
        buffer[j] = buffer[i - j - 1];
        buffer[i - j - 1] = temp;
    }
}

__device__ int stringLength(const char* str) {
    int length = 0;
    while (str[length] != '\0') {
        length++;
    }
    return length;
}

__device__ void stringConcat(char* dest, const char* src) {
    int destLen = stringLength(dest);
    int i = 0;
    while (src[i] != '\0') {
        dest[destLen + i] = src[i];
        i++;
    }
    dest[destLen + i] = '\0';
}

__device__ void timeToString(time_t value, char* buffer) {
    uint32ToString(static_cast<uint32_t>(value), buffer);
}

__device__ bool compareStrings(const char* str1, const char* str2) {
    while (*str1 && (*str1 == *str2)) {
        ++str1;
        ++str2;
    }
    return (*str1 - *str2) == 0;
}

__device__ void myStrcpy(char* dest, const char* src) {
    while (*src) {
        *dest++ = *src++;
    }
    *dest = '\0';
}

__device__ void mySubstr(const char* input, char* output, int start, int length) {
    int i;
    for (i = 0; i < length && input[start + i] != '\0'; ++i) {
        output[i] = input[start + i];
    }
    output[i] = '\0';
}

__global__ void MineKernel(uint32_t nDifficulty, const char* str, uint32_t _nIndex, const char* sPrevHash, time_t _tTime, const char* _sData, uint32_t* resultNonce, char* resultHash) {
    __shared__ int found;
    if (threadIdx.x == 0) {
        found = 0;
    }
    __syncthreads();
    
    if(atomicAdd(&found, 0) == 1) return; // Se um hash válido foi encontrado, retorne (saia do kernel)

    // Calculate the thread's unique nonce
    uint32_t _nNonce = blockIdx.x * blockDim.x + threadIdx.x;

    // Prepare the string to be hashed
    char indexBuffer[11]; // buffer para _nIndex (máximo 10 dígitos + \0)
    char timeBuffer[21];  // buffer para _tTime (máximo 20 dígitos + \0)
    char nonceBuffer[11]; // buffer para _nNonce (máximo 10 dígitos + \0)
    uint32ToString(_nIndex, indexBuffer);
    timeToString(_tTime, timeBuffer);
    uint32ToString(_nNonce, nonceBuffer);

    int bufferSize = stringLength(indexBuffer) + stringLength(sPrevHash) + stringLength(timeBuffer) + stringLength(_sData) + stringLength(nonceBuffer);

    // Use um buffer local em vez de alocação dinâmica
    char outputBuffer[256];
    if (bufferSize + 1 > sizeof(outputBuffer)) {
        return; // Evita estouro de buffer
    }
    outputBuffer[0] = '\0';

    stringConcat(outputBuffer, indexBuffer);
    stringConcat(outputBuffer, sPrevHash);
    stringConcat(outputBuffer, timeBuffer);
    stringConcat(outputBuffer, _sData);
    stringConcat(outputBuffer, nonceBuffer);

    // Perform the hash calculation
    char* sHash = sha256(outputBuffer);

    // Check if the hash meets the difficulty criteria
    char tempStr[65];
    mySubstr(sHash, tempStr, 0, nDifficulty);
    if (compareStrings(tempStr, str)) {
        atomicExch(&validHashFound, 1); // Set the flag using an atomic exchange
        if (atomicExch(&found, 1) == 0) {
            *resultNonce = _nNonce;
            myStrcpy(resultHash, sHash);
        }
    }
}

void Block::MineBlock(uint32_t nDifficulty)
{
    // Prepare the difficulty string
    char cstr[nDifficulty + 1];
    for (uint32_t i = 0; i < nDifficulty; ++i)
    {
        cstr[i] = '0';
    }
    cstr[nDifficulty] = '\0';

    // Allocate memory on the GPU for the result nonce and hash
    uint32_t* d_resultNonce;
    char* d_resultHash;
    hipMalloc(&d_resultNonce, sizeof(uint32_t));
    hipMalloc(&d_resultHash, 65); // SHA256 hash is 64 characters + null terminator

    // Reset the validHashFound flag to false
    unsigned int flag = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(validHashFound), &flag, sizeof(unsigned int));

    // Copy the block data to the GPU
    uint32_t h_nIndex = _nIndex;
    time_t h_tTime = _tTime;
    char* d_sPrevHash;
    char* d_sData;
    hipMalloc(&d_sPrevHash, sPrevHash.length() + 1);
    hipMalloc(&d_sData, _sData.length() + 1);

    hipMemcpy(d_sPrevHash, sPrevHash.c_str(), sPrevHash.length() + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_sData, _sData.c_str(), _sData.length() + 1, hipMemcpyHostToDevice);

    // Calculate the number of blocks and threads
    uint32_t numThreads = NUM_THREADS;
    uint32_t numBlocks = (MAX_NONCE + numThreads - 1) / numThreads;

    // Launch the kernel
    MineKernel<<<numBlocks, numThreads>>>(nDifficulty, cstr, h_nIndex, d_sPrevHash, h_tTime, d_sData, d_resultNonce, d_resultHash);

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Check for any errors during kernel execution
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        // Print the CUDA error message and exit
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Check if a valid hash was found
    uint32_t resultNonce;
    char resultHash[65];
    hipMemcpy(&resultNonce, d_resultNonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(resultHash, d_resultHash, 65, hipMemcpyDeviceToHost);

    if (resultNonce != 0) // Assuming 0 is the initial value and means no valid nonce was found
    {
        std::cout << "Block mined: " << resultHash << std::endl;
    }
    else
    {
        std::cout << "No valid hash found within the nonce range." << std::endl;
    }

    // Free the GPU memory
    hipFree(d_resultNonce);
    hipFree(d_resultHash);
    hipFree(d_sPrevHash);
    hipFree(d_sData);
}

inline std::string Block::_CalculateHash() const
{
    std::stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << _nNonce;

    return sha256host(ss.str());
}
